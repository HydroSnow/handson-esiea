#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to 
 * initialize an array to a specific value, after which the array is 
 * copied to the host (CPU) memory.  To increase performance, multiple 
 * kernel/memcopy pairs are launched asynchronously, each pair in its 
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <stdio.h>
#include <stdlib.h>


__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for(int i=0;i<num_iterations;i++)
    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
  for(int i = 0; i < n; i++)
    if(a[i] != c)
    {
			printf("%d: %d %d\n", i, a[i], c);
      return 0;
		}
  return 1;
}

int main(int argc, char *argv[])
{
  int cuda_device = 0;
  int nstreams = 4;               // number of streams for CUDA calls
  int nreps = 10;                 // number of times each experiment is repeated
  int n = 16 * 1024 * 1024;       // number of ints in the data set
  int nbytes = n * sizeof(int);   // number of data bytes
  dim3 threads, blocks;           // kernel launch configuration
  float elapsed_time;             // timing variables
  int niterations;	              // number of iterations for the loop inside the kernel
  
  if( argc > 1 )
    cuda_device = atoi( argv[1] );
  
#ifdef __DEVICE_EMULATION__
  n = 4096;   // reduced workload for emulation (n should be divisible by 512*nstreams)
#endif
  
  // check the compute capability of the device
  int num_devices=0;
  checkCudaErrors(hipGetDeviceCount(&num_devices));
  if(0==num_devices)
  {
    printf("your system does not have a CUDA capable device\n");
    return 1;
  }
	
  // check if the command-line chosen device ID is within range, exit if not
  if( cuda_device >= num_devices )
  {
    printf("choose device ID between 0 and %d\n", num_devices-1);
    return 1;
  }
  
  hipSetDevice( cuda_device );
  
  hipDeviceProp_t device_properties;
	
  checkCudaErrors( hipGetDeviceProperties(&device_properties, cuda_device) );
  if( (1 == device_properties.major) && (device_properties.minor < 1))
    printf("%s does not have compute capability 1.1 or later\n\n", device_properties.name);
  if(device_properties.minor > 1)
    niterations = 5;
  else
    niterations = 2; // reduced workload for compute capability 1.0 and 1.1
  
  printf("running on: %s with compute capability %d.%d\n", device_properties.name, device_properties.major, device_properties.minor);
  
  // allocate host memory
  int c = 5; // value to which the array will be initialized  
  int * a = 0; // pointer to the data array in memory
  
  /*
   * TODO : change allocation of array a to use pinned memory
   */
  a = (int *) malloc(nbytes);

  // allocate device memory
  int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
  checkCudaErrors( hipMalloc(&d_a, nbytes) );
  checkCudaErrors( hipMalloc(&d_c, sizeof(int)) );
  checkCudaErrors( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice) );
  
  /*
   * TODO :  allocate and initialize an array of stream handles
   */
  hipStream_t *streams = ...;
  for(int i = 0; i < nstreams; i++)
    // TODO
  
  // create CUDA event handles
  hipEvent_t start_event, stop_event;
  checkCudaErrors( hipEventCreate(&start_event) );
  checkCudaErrors( hipEventCreate(&stop_event) );

  //////////////////////////////////////////////////////////////////////
  // time non-streamed execution for reference
  //////////////////////////////////////////////////////////////////////
  threads=dim3(512, 1);
  blocks=dim3(n / threads.x, 1);
  hipEventRecord(start_event, 0);
  for(int k = 0; k < nreps; k++)
  {
    init_array<<<blocks, threads>>>(d_a, d_c, niterations);
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
  }
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  checkCudaErrors( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
  printf("non-streamed:\t%.2f ms\n", elapsed_time / nreps);
  
  //////////////////////////////////////////////////////////////////////
  // time execution with nstreams streams
  //////////////////////////////////////////////////////////////////////
  threads=dim3(512,1);

  /*
   * TODO : modify the grid dimensions according to the streams configuration
   */
  blocks = dim3(n / threads.x, 1);
   
  memset(a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
  hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
  hipEventRecord(start_event, 0);
  for(int k = 0; k < nreps; k++)
  {
    /*
     * TODO : launch execution of nstreams streams
     */
   
    
    /*
     * TODO : use the asynchronous copy function to retrieve data from the device into nstreams streams
     */
    
  }
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  checkCudaErrors( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
  printf("%d streams:\t%.2f ms\n", nstreams, elapsed_time / nreps);

  // check whether the output is correct
  printf("-------------------------------\n");
  if(correct_data(a, n, c*nreps*niterations))	// each element should have been incremented by c a total of npres*niterations times
    printf("Test PASSED\n");
  else
    printf("Test FAILED\n");
  
  // release resources
  /*
   * TODO : destroy the nstreams streams
   */ 
  for(int i = 0; i < nstreams; i++)
    // TODO

  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
  
  /*
   * TODO : free the pinned memory allocated for array a
   */


  hipFree(d_a);
  hipFree(d_c);

  return 0;
}
